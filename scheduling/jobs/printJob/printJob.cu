#include "hip/hip_runtime.h"

#include "printJob.h"

// callback that is envoked at the end of each kernel execution.
void CUDART_CB PrintJob::printKernelCallback(hipStream_t stream,
                                             hipError_t status, void *data) {

  // get the kernel launch config that has to be cleaned up and that contains
  // info to display.
  printKernelLaunchInformation *kernelInfo =
      static_cast<printKernelLaunchInformation *>(data);

  // copy the result from device to host.
  hipMemcpy(kernelInfo->hostPtr, kernelInfo->devicePtr, kernelInfo->size,
             hipMemcpyDeviceToHost);

  // std::cout<<"print job from task "<<kernelInfo->taskId<<" took
  // "<<*(kernelInfo->hostPtr)<<"s\n";

  // free the dynamically allocated memory and the stream.
  free(kernelInfo->hostPtr);
  hipFree(kernelInfo->devicePtr);
  hipStreamDestroy(stream);
  // std::cout << "print job finished\n";

  float currentTime = getCurrentTime();
  // notify the scheduler the job is done executing.
  Job::notifyJobCompletion(kernelInfo->jobPtr, currentTime);
}

// callback constructor.
void PrintJob::addPrintKernelCallback(Job *job, hipStream_t stream,
                                      float *dptr, float *hptr, size_t size,
                                      int id) {

  printKernelLaunchInformation *kernelInfo =
      new printKernelLaunchInformation(job, stream, dptr, hptr, size, id);
  hipStreamAddCallback(stream, printKernelCallback, kernelInfo, 0);
}

// job execute function.
void PrintJob::execute() {
  hipStream_t kernel_stream;
  hipStreamCreate(&kernel_stream);
  float *d_output;
  float *h_output = (float *)std::malloc(sizeof(float));
  hipMalloc(&d_output, sizeof(float));

  printMessage<<<1, 1, 0, kernel_stream>>>(1, 1, 100, d_output);
  addPrintKernelCallback(this, kernel_stream, d_output, h_output, sizeof(float),
                         1);

  return;
}

PrintJob::PrintJob(int threadsPerBlock, int threadBlocks) {
  this->threadsPerBlock = threadsPerBlock;
  this->threadBlocks = threadBlocks;

  int totalThreads = threadsPerBlock * threadBlocks;
  int neededSMs =
      totalThreads / DeviceInfo::getDeviceProps()->getMaxThreadsPerSM();

  if (neededSMs < 1) {
    this->neededTPCs = 1;
    return;
  }
  this->neededTPCs =
      int(ceil(neededSMs / DeviceInfo::getDeviceProps()->getSMsPerTPC()));
}
