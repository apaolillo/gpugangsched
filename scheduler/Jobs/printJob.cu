#include <iostream>
#include <hip/hip_runtime.h>






__global__ static void printMessage(int taskId, int jobId){
  printf("hello from task: %d and job: %d", taskId, jobId);
}



void executeJob(int taskId, int jobId){
  printMessage<<<1, 1>>>(taskId, jobId);

}
