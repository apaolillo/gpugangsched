   
#include "busyJob.h"
    //callback that is envoked at the end of each kernel execution.
void CUDART_CB BusyJob::busyKernelCallback(hipStream_t stream, hipError_t status, void *data){

  //get the kernel launch config that has to be cleaned up and that contains info to display.
  BusyJob::busyKernelLaunchInformation* kernelInfo = static_cast<BusyJob::busyKernelLaunchInformation*>(data);
  
  //copy the result from device to host.
  hipMemcpy(kernelInfo->hostPtr, kernelInfo->devicePtr, kernelInfo->size, hipMemcpyDeviceToHost);

  std::cout<<"busy job from task "<<kernelInfo->taskId<<" took "<<*(kernelInfo->hostPtr)<<"s\n";
  
  //free the dynamically allocated memory and the stream.
  free(kernelInfo->hostPtr);
  hipFree(kernelInfo->devicePtr);
  hipFree(kernelInfo->timerDptr);
  hipStreamDestroy(stream);

}   


//callback constructor.
void BusyJob::addBusyKernelCallback(hipStream_t stream, float* dptr, float* timerDptr, float* hptr, size_t size, int id){

  BusyJob::busyKernelLaunchInformation* kernelInfo = new BusyJob::busyKernelLaunchInformation(stream, dptr, timerDptr, hptr, size, id);

  hipStreamAddCallback(stream, busyKernelCallback, kernelInfo, 0);

}



  

//job definition that goes with a task.
void BusyJob::execute(){
  // Get device hipGetDeviceProperties
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
    

    
  // Allocate memory
  float *d_output, *d_timer;
  hipMalloc(&d_output, sizeof(float));
  hipMalloc(&d_timer, sizeof(float));

  hipStream_t kernel_stream;
  hipStreamCreate(&kernel_stream);

  float *h_output = (float*)std::malloc(sizeof(float));
  
  

  maxUtilizationKernel<<<1, 1, 0, kernel_stream>>>(d_output, d_timer, 1000);
  addBusyKernelCallback(kernel_stream, d_output, d_timer, h_output, sizeof(float), 1);

  return;

} 



BusyJob::BusyJob(int minimumTPCs, int maximumTPCs){
  this->maximumTPCs = maximumTPCs;
  this->minimumTPCs = minimumTPCs;
} 







