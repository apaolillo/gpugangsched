#include <iostream>
#include <hip/hip_runtime.h>
#include "kernels/busyKernel.cu"
#include "job.h"



template<typename... FuncArgs>
class BusyJob: public Job{
  private:
    int minimumTPCs, maximumTPCs;
    float releaseTime, maximalExecutionTime, absoluteDeadline;





    struct busyKernelLaunchInformation {
      hipStream_t kernelStream; //stream in which the kernel is launched. 
      float*       devicePtr;    // Device memory pointer
      float*       timerDptr;    //device ptr to float that holds the total execution time of a kernel.
      float*       hostPtr;      // Host memory pointer
      size_t       size;         // Size of data to copy in bytes
      int          taskId;       //id of  the task invoking jobs.
    

      busyKernelLaunchInformation(hipStream_t stream, float* dptr, float* timeDptr, float* hptr, size_t sz, int id)
        : kernelStream(stream), devicePtr(dptr), timerDptr(), hostPtr(hptr), size(sz), taskId(id){}

    };



    //callback that is envoked at the end of each kernel execution.
    static void CUDART_CB busyKernelCallback(hipStream_t stream, hipError_t status, void *data){

      //get the kernel launch config that has to be cleaned up and that contains info to display.
      busyKernelLaunchInformation* kernelInfo = static_cast<busyKernelLaunchInformation*>(data);
  
      //copy the result from device to host.
      hipMemcpy(kernelInfo->hostPtr, kernelInfo->devicePtr, kernelInfo->size, hipMemcpyDeviceToHost);

      std::cout<<"busy job from task "<<kernelInfo->taskId<<" took "<<*(kernelInfo->hostPtr)<<"s\n";
  
      //free the dynamically allocated memory and the stream.
      free(kernelInfo->hostPtr);
      hipFree(kernelInfo->devicePtr);
      hipFree(kernelInfo->timerDptr);
      hipStreamDestroy(stream);

    }   


    //callback constructor.
    static void addBusyKernelCallback(hipStream_t stream, float* dptr, float* timerDptr, float* hptr, size_t size, int id){

      busyKernelLaunchInformation* kernelInfo = new busyKernelLaunchInformation(stream, dptr, timerDptr, hptr, size, id);

      hipStreamAddCallback(stream, busyKernelCallback, kernelInfo, 0);

    }



  public:

    //job definition that goes with a task.
    static void executeJob(int taskId, int jobId, int loopDuration){
      // Get device hipGetDeviceProperties
      hipDeviceProp_t deviceProp;
      hipGetDeviceProperties(&deviceProp, 0);
    

    
      // Allocate memory
      float *d_output, *d_timer;
      hipMalloc(&d_output, sizeof(float));
      hipMalloc(&d_timer, sizeof(float));

      hipStream_t kernel_stream;
      hipStreamCreate(&kernel_stream);

      float *h_output = (float*)std::malloc(sizeof(float));
  
  

      maxUtilizationKernel<<<1, 1, 0, kernel_stream>>>(d_output, d_timer, loopDuration);
      addBusyKernelCallback(kernel_stream, d_output, d_timer, h_output, sizeof(float), taskId);

      return;

    } 



    BusyJob(int minimumTPCs, int maximumTPCs): minimumTPCs(minimumTPCs), maximumTPCs(maximumTPCs)
    {} 




    void launchJob() override{
      auto& args_tuple = std::any_cast<std::tuple<FuncArgs...>&>(args);
      std::apply(executeJob, args_tuple);
    }

};




