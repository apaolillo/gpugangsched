#include <iostream>
#include <hip/hip_runtime.h>
#include <functional>
#include <tuple>
#include <utility>
#include <any>
/**
 * Representation of a task. Includes all the necessary elements of a PERIODIC task.
 *
 */


class TaskBase{
  public:
    virtual void launchJob() = 0;


    template<typename... Args>
    void execute(Args&&... args){
      this->args = std::make_tuple(std::forward<Args>(args)...);
      launchJob();
    }


    virtual ~TaskBase() = default;

  protected:
    std::any args;


};


template<typename... FuncArgs>
class Task: public TaskBase{
  private:
    int offset, compute_time, rel_deadline, period, id;
    
    //function that does not return anything and takes any amount of args with any type. 
    std::function<void(FuncArgs...)> job;
    

  public:

    Task(int offset, int compute_time, int rel_deadline, int period, std::function<void(FuncArgs...)>job, int id)
      :offset(offset), compute_time(compute_time), rel_deadline(rel_deadline), period(period), job(job), id(id) {}
    
    void launchJob() override{
      auto& args_tuple = std::any_cast<std::tuple<FuncArgs...>&>(args);
      std::apply(job, args_tuple);
    }

    int get_offset(){
      return this->offset;
    }

    int get_compute_time(){
      return this->compute_time;
    }

    int get_rel_deadline(){
      return this->rel_deadline;
    }

    int get_period(){
      return this->period;
    }
   
};
