#include <iostream>
#include <hip/hip_runtime.h>
#include <functional>
#include <iterator>
#include <tuple>
#include <utility>
#include <any>
#include <chrono>
/**
 * Representation of a task. Includes all the necessary elements of a PERIODIC task.
 *
 */


class TaskBase{
  public:
    virtual void launchJob() = 0;


    template<typename... Args>
    void execute(Args&&... args){
      this->args = std::make_tuple(std::forward<Args>(args)...);
      launchJob();
    }


    virtual ~TaskBase() = default;

  protected:
    std::any args;


};


template<typename... FuncArgs>
class Task: public TaskBase{
  private:
    int offset, compute_time, rel_deadline, period, id;
    std::chrono::system_clock::time_point beginTime, currentTime;
    //function that does not return anything and takes any amount of args with any type. 
    std::function<void(FuncArgs...)> job;

    bool firstJobReleased = false;
    

  public:

    Task(int offset, int compute_time, int rel_deadline, int period, std::function<void(FuncArgs...)>job, int id, std::chrono::system_clock::time_point beginTime, std::chrono::system_clock::time_point currentTime)
      :offset(offset), compute_time(compute_time), rel_deadline(rel_deadline), period(period), job(job), id(id), beginTime(beginTime), currentTime(currentTime) {}
    
    void launchJob() override{
      if(!firstJobReleased){
        firstJobReleased = true;
      }
      auto& args_tuple = std::any_cast<std::tuple<FuncArgs...>&>(args);
      std::apply(job, args_tuple);
    }

    int get_offset(){
      return this->offset;
    }

    int get_compute_time(){
      return this->compute_time;
    }

    int get_rel_deadline(){
      return this->rel_deadline;
    }

    int get_period(){
      return this->period;
    }
   
};
