#include <iostream>
#include <hip/hip_runtime.h>
#include <memory>
#include "Tasks/task.h"
#include "Jobs/printJob.h"
#include "Jobs/busyJob.h"
#include "Jobs/jobFactory.h"
#include "common/helpFunctions.h"
#include "schedulers/JLFP.h"

int main(){
  
  
  std::vector<Task*> tasks;

  auto printJobFactory = TemplatedJobFactoryHelper<PrintJob, int, int, int>::create(10, 10); 
  auto busyJobFactory = TemplatedJobFactoryHelper<BusyJob, int, int, int>::create(10, 10);

  Task task1(10, 5, 20, 100, std::move(printJobFactory), 1);
  Task task2(10, 5, 20, 100, std::move(busyJobFactory), 2);

  tasks.push_back(&task1);
  tasks.push_back(&task2);

  JLFP scheduler1;
  
  while(true){

    for(Task* task : tasks){
      if(task->isJobReady()){
        scheduler1.addJob(task->releaseJob());
      }
    }
    scheduler1.displayQueueJobs();
    scheduler1.dispatch();
    sleep(2000);

  }



  return 0;
}

