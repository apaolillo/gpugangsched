#include "hip/hip_runtime.h"
// This file defines a CUDA benchmark which issues multiple kernels to a single
// stream before waiting for all kernels to complete. The configuration for the
// kernels is taken exclusively from the additional_info field in the
// InitializationParameters struct. The actual kernels will simply be instances
// of the same kernel as in the timer_spin benchmark. This benchmark ignores
// all fields in its initialization parameters apart from cuda_device and
// additional_info.
//
// The format of the necessary additional_info field is as follows. Each object
// in the "actions" list must have a type that is one of "kernel", "malloc",
// "free", "memset", "memcpy", or "synchronize". Memory operations such as
// malloc, free memset, and memcpy operate on buffers separate from each other.
// For example, a malloc doesn't need to precede a memset, because memset
// buffers will be allocated during initialization. The only limitation is that
// only a small number of unbalanced malloc and free operations are allowed.
// Any unfreed mallocs from these actions will be freed during benchmark
// cleanup. Synchronization actions are available solely to experiment with
// scheduling, and are not necessary for the task. A stream-synchronization
// request will be issued at the end of all actions regardless of whether an
// explicit, additional synchronization action was carried out.

// For more details about parameters for each action, see the annotated JSON
// structure below:
/*
"additional_info": {
  "use_null_stream": <Boolean, defaults to false, set to true to use the
    null stream rather than the default stream>,
  "actions": [
    {
      "delay": <A floating-point number of seconds to sleep before starting
        this action. Defaults to 0.0, which will insert no sleep at all.>,
      "type": <A string, from the list given above.>,
      "label": <A string, a label for this action.>,
      "parameters": <A JSON object with action-specific parameters.>
    },
    {
      "type": "kernel",
      "label": "Kernel 1",
      "parameters": {
        "type": <A string: "timer_spin" or "counter_spin". Defaults to
          "timer_spin">,
        "duration": <If "type" is "timer_spin", this will be the number of
          nanoseconds to run the kernel. If type is "counter_spin", this
          will be the number of loop iterations to run.>,
        "shared_memory_size": <The number of shared 32-bit integers to use.
          Defaults to 0. Must be 0, 4096, 8192, or 10240.>,
        "block_count": <The number of thread blocks to use. Defaults to the
          value given in the benchmark parameters.>,
        "thread_count": <The number of threads per block to use. Defaults to
          the value given in the benchmark parameters.>
      },
    }
    {
      "type": "malloc",
      "label": "Malloc 1",
      "parameters": {
        "host": <Boolean. Defaults to false. If true, will allocate host
          memory.>,
        "size": <Number of bytes to allocate>
      }
    },
    {
      "type": "free",
      "label": "Free 1",
      "parameters": {
        "host": <Boolean. Defaults to false. If true, will free host memory.
          The entire "parameters" block can be omitted here for the default.>
      }
    },
    {
      "type": "memset",
      "label": "Memset 1",
      "parameters": {
        "async": <Boolean. Defaults to true. If false, will issue a
          null-stream memset regardless of use_null_stream's value.>,
        "size": <Number of bytes to set to 0>
      }
    },
    {
      "type": "memcpy",
      "label": Memcpy 1",
      "parameters": {
        "async": <Boolean. Defaults to true. If false, issues a null-stream
          memcpy regardless of use_null_stream's value.>,
        "size": <Number of byte to copy>,
        "direction": <Either "deviceToDevice", "deviceToHost", or
          "hostToDevice">
      }
    },
    {
      "type": "synchronize",
      "label": "Sync 1",
      "parameters": {
        "device": <Boolean. Defaults to false (parameters can be omitted here
          entirely, too). If true, runs a hipDeviceSynchronize rather than
          hipStreamSynchronize.>
      }
    }
  ]
}
*/
// Actions are issued to the stream in the same order that they're specified
// in the "actions" list.
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include "benchmark_gpu_utilities.h"
#include "library_interface.h"
#include "third_party/cJSON.h"

// This specifies the maximum number of un-freed malloc actions that can occur
// before further allocations return an error instead. Any list with this many
// or fewer (balanced) malloc and free actions can run indefinitely.
#define MAX_MEMORY_ALLOCATION_COUNT (10)

// This specifies the number of pre-allocated buffers that are allocated during
// initialization, so that a number of free actions can be used without a
// preceding malloc. This can be at most MAX_MEMORY_ALLOCATION_COUNT.
#define INITIAL_ALLOCATION_COUNT (4)

// This speicifies the size, in bytes, of the pre-allocated buffers.
#define INITIAL_ALLOCATION_SIZE (1024)

// This macro is used to create functions that statically use predefined
// amounts of shared memory. This is used by the GENERATE_KERNEL macro.
#define GENERATE_SHARED_MEMORY_FUNCTION(amount) \
  static __device__ uint32_t UseSharedMemory_##amount(void) { \
    __shared__ uint32_t shared_array[(amount)]; \
    uint32_t elements_per_thread, i; \
    elements_per_thread = (amount) / blockDim.x; \
    for (i = 0; i < elements_per_thread; i++) { \
      shared_array[threadIdx.x * elements_per_thread + i] = threadIdx.x; \
    } \
    return shared_array[threadIdx.x * elements_per_thread]; \
  }

// Generates kernels that use the given amount of shared memory. Kernels have
// names like SharedMemGPUSpin_<amount>, and take the following parameters:
// (int counter, uint64_t duration, uint64_t *block_times,
// uint32_t *block_smids, uint64_t *junk). If the "counter" parameter is
// nonzero, then a constant amount of computation will be carried out rather
// than waiting for a constant amount of time. The "junk" parameter is used to
// prevent optimizations, and must be NULL. Otherwise, this kernel operates
// similarly to the simpler GPUSpin kernel in stream_action.cu. This WILL NOT
// work for 0 bytes of shared memory--that's what the plain GPUSpin in
// stream_action.cu is for.
#define GENERATE_SPIN_KERNEL(amount) \
  /* Produce a function that uses shared memory */ \
  GENERATE_SHARED_MEMORY_FUNCTION(amount) \
  static __global__ void SharedMemGPUSpin_##amount(int use_counter, \
    uint64_t duration, uint64_t *block_times, uint32_t *block_smids, \
    uint64_t *junk) { \
    uint32_t shared_mem_res; \
    uint64_t i, accumulator; \
    uint64_t start_time = GlobalTimer64(); \
    if (threadIdx.x == 0) { \
      block_times[blockIdx.x * 2] = start_time; \
      block_smids[blockIdx.x] = GetSMID(); \
    } \
    __syncthreads(); \
    /* shared_mem_res is our thread index */ \
    shared_mem_res = UseSharedMemory_##amount(); \
    if (use_counter) { \
      for (i = 0; i < duration; i++) { \
        accumulator += i; \
      } \
    } else { \
      while ((GlobalTimer64() - start_time) < duration) { \
        continue; \
      } \
    } \
    if (junk) *junk = accumulator; \
    if (shared_mem_res == 0) { \
      block_times[blockIdx.x * 2 + 1] = GlobalTimer64(); \
    } \
  }

// This holds parameters for the kernel action.
typedef struct {
  // The grid dimensions for this kernel.
  int block_count;
  int thread_count;
  // The amount of shared memory used by this kernel.
  int shared_memory_count;
  // If this is nonzero, the counter_spin kernel will be used, which performs
  // a constant amount of busywork computations. If this is zero, the
  // timer_spin kernel will be used instead, which waits until a certain number
  // of nanoseconds have elapsed.
  int use_counter_spin;
  // The number of either spin iterations or nanoseconds this kernel runs for
  // (depending on whether it is a timer spin or counter spin kernel).
  uint64_t duration;
  // Hold the times needed for a CUDA kernel.
  uint64_t *device_block_times;
  uint64_t *host_block_times;
  uint32_t *device_smids;
  uint32_t *host_smids;
} KernelParameters;

// This holds parameters for the hipMalloc action.
typedef struct {
  // This is the number of bytes to allocate.
  uint64_t size;
  // If nonzero, call hipHostMalloc rather than hipMalloc.
  int allocate_host_memory;
} MallocParameters;

// This holds parameters for the hipFree action.
typedef struct {
  // If nonzero, call hipHostFree rather than hipFree.
  int free_host_memory;
} FreeParameters;

// This holds parameters for the hipMemset action, which sets bytes to a
// random 8-bit value.
typedef struct {
  // If nonzero, then hipMemset will be called (associated with no stream),
  // rather than hipMemsetAsync, which will use the task's specified stream.
  int synchronous;
  // This contains the number of bytes to set.
  uint64_t size;
} MemsetParameters;

// This holds parameters for the hipMemcpy action, which copies data between
// host and device, or two device buffers.
typedef struct {
  // One of the hipMemcpyKind values. However, values 0 (host - host) and 4
  // (unspecified) are not supported.
  hipMemcpyKind direction;
  // If nonzero, then hipMemcpy will be used. If 0, then hipMemcpyAsync is
  // used, associated with the task's stream.
  int synchronous;
  // The number of bytes to copy.
  uint64_t size;
} MemcpyParameters;

// This holds parameters for the synchronize action.
typedef struct {
  // If this is nonzero, then hipDeviceSynchronize will be called. Otherwise,
  // hipStreamSynchronize is called, associated with the task's stream.
  int sync_device;
} SyncParameters;

// This is used as a tag to identify the parameters and behavior to carry out
// for each action supported by the benchmark.
typedef enum {
  ACTION_UNINITIALIZED = 0,
  ACTION_KERNEL,
  ACTION_MALLOC,
  ACTION_FREE,
  ACTION_MEMSET,
  ACTION_MEMCPY,
  ACTION_SYNC,
} ActionType;

// This defines the behavior and parameters for all potential actions.
typedef struct {
  // The number of seconds to sleep after the current action's completion,
  // before launching this one.
  double delay;
  // The label (typically a kernel name) to give this action.
  char *label;
  ActionType type;
  union {
    KernelParameters kernel;
    MallocParameters malloc;
    FreeParameters free;
    MemsetParameters memset;
    MemcpyParameters memcpy;
    SyncParameters sync;
  } parameters;
} ActionConfig;

// Holds local information for each instantiation of this benchmark.
typedef struct {
  // The CUDA stream with which all operations will be associated.
  hipStream_t stream;
  // The CUDA stream with which copy_out operations will be associated. May
  // differ from the regular stream, because this will never be the NULL
  // stream.
  hipStream_t copy_out_stream;
  // This will be set to 1 if the stream was created and must be closed during
  // cleanup (it can remain 0 if the NULL stream is used).
  int stream_created;
  // The number of actions to perform per execution.
  int action_count;
  // The list of actions to perform.
  ActionConfig *actions;
  // The number of actions which are kernel launches.
  int kernel_count;
  // Information to provide to the host process about block start and end times
  // for each kernel action.
  KernelTimes *kernel_times;
  // A buffer of host memory for copies and memsets. May be NULL if not needed.
  // Is guaranteed to be the size of the largest copy or memset needed by any
  // action.
  uint8_t *host_copy_buffer;
  // A buffer of device memory for copies and memsets. May be NULL if not
  // needed. This is guaranteed to be the size of the largest copy or memset
  // needed by any action.
  uint8_t *device_copy_buffer;
  // This will be a secondary device buffer, but will only be allocated if a
  // device-to-device memory copy is used.
  uint8_t *device_secondary_buffer;
  // This is a stack of pointers to device memory allocated by hipMalloc
  // actions.
  uint8_t **device_memory_allocations;
  // Holds the number of pointers in the device_memory_allocations list. This
  // increases with each hipMalloc action and decreases with each hipFree.
  int device_memory_allocation_count;
  // This is a stack of pointers to host memory allocated by hipHostMalloc.
  // It works in the same way as device_memory_allocations.
  uint8_t **host_memory_allocations;
  // This is analagous to device_memory_allocation_count, but for host memory
  // allocations.
  int host_memory_allocation_count;
} TaskState;

// Use the macros defined in stream_action.h to generate a set of kernels using
// various amounts of static shared memory.
GENERATE_SPIN_KERNEL(4096);
GENERATE_SPIN_KERNEL(8192);
GENERATE_SPIN_KERNEL(10240);

// A basic kernel that wastes GPU cycles without using shared memory. The
// duration parameter specifies the number of nanoseconds to wait if
// use_counter is 0. If use_counter is nonzero, duration specifies a number of
// loop iterations to spin instead. The junk parameter must be NULL and is used
// to prevent optimization.
static __global__ void GPUSpin(int use_counter, uint64_t duration,
    uint64_t *block_times, uint32_t *block_smids, uint64_t *junk) {
  uint64_t i, accumulator;
  uint64_t start_time = GlobalTimer64();
  // Have one thread record the block's start time and SM ID.
  if (threadIdx.x == 0) {
    block_times[blockIdx.x * 2] = start_time;
    block_smids[blockIdx.x] = GetSMID();
  }
  __syncthreads();
  if (use_counter) {
    // Write to the accumulator (which must be potentially returned) to prevent
    // this loop from being optimized out.
    for (i = 0; i < duration; i++) {
      accumulator += i;
    }
  } else {
    // Wait until the specified number of nanoseconds has elapsed.
    while ((GlobalTimer64() - start_time) < duration) {
      continue;
    }
  }
  // Make it look like the junk value can be used to prevent the loop updating
  // the accumulator from being removed by the optimizer.
  if (junk) *junk = accumulator;
  // Have one thread write the block end time (simple, but may be slightly
  // inaccurate if other warps finish later).
  if (threadIdx.x == 0) {
    block_times[blockIdx.x * 2 + 1] = GlobalTimer64();
  }
}

// Frees any data and clears out an ActionConfig struct. For use during
// cleanup.
static void CleanupAction(ActionConfig *action) {
  uint64_t *tmp64;
  uint32_t *tmp32;
  if (action->label) free(action->label);
  if (action->type == ACTION_KERNEL) {
    // For now, only kernel actions require extra cleanup.
    tmp64 = action->parameters.kernel.device_block_times;
    if (tmp64) CheckCUDAError(hipFree(tmp64));
    tmp64 = action->parameters.kernel.host_block_times;
    if (tmp64) CheckCUDAError(hipHostFree(tmp64));
    tmp32 = action->parameters.kernel.device_smids;
    if (tmp32) CheckCUDAError(hipFree(tmp32));
    tmp32 = action->parameters.kernel.host_smids;
    if (tmp32) CheckCUDAError(hipHostFree(tmp32));
  }
  memset(action, 0, sizeof(*action));
}

// Implements the cleanup fucntion required by the interface, but is also used
// internally to clean up during a faulty Initialize(). That's why all of the
// pointers are checked to be non-NULL. This is also why it's very important to
// ensure that any fields and pointers are zero before any initialization.
static void Cleanup(void *data) {
  TaskState *state = (TaskState *) data;
  int i;
  ActionConfig *action = NULL;
  for (i = 0; i < state->action_count; i++) {
    action = state->actions + i;
    CleanupAction(action);
  }
  if (state->actions) free(state->actions);
  if (state->kernel_times) free(state->kernel_times);
  // The CheckCUDAError macros here are just to print a message on error, since
  // we can't really do any additional error handling during cleanup.
  if (state->stream_created) {
    // Remember that state->stream may be the NULL stream or may be another
    // reference to this same stream. In either case, we don't need to destroy
    // it.
    CheckCUDAError(hipStreamDestroy(state->copy_out_stream));
  }
  if (state->host_copy_buffer) {
    CheckCUDAError(hipHostFree(state->host_copy_buffer));
  }
  if (state->device_copy_buffer) {
    CheckCUDAError(hipFree(state->device_copy_buffer));
  }
  if (state->device_secondary_buffer) {
    CheckCUDAError(hipFree(state->device_secondary_buffer));
  }
  for (i = 0; i < state->device_memory_allocation_count; i++) {
    CheckCUDAError(hipFree(state->device_memory_allocations[i]));
  }
  if (state->device_memory_allocations) free(state->device_memory_allocations);
  for (i = 0; i < state->host_memory_allocation_count; i++) {
    CheckCUDAError(hipHostFree(state->host_memory_allocations[i]));
  }
  if (state->host_memory_allocations) free(state->host_memory_allocations);
  memset(state, 0, sizeof(*state));
  free(state);
}

// Returns nonzero if all of the keys in the JSON object are in the list of
// valid keys.
static int VerifyJSONKeys(cJSON *object, const char* const valid_keys[],
    int valid_count) {
  int i, found;
  // We'll be passed a top-level object here.
  object = object->child;
  while (object != NULL) {
    found = 0;
    if (!object->string) {
      printf("Found JSON object without a name in stream_action settings.\n");
      return 0;
    }
    for (i = 0; i < valid_count; i++) {
      if (strcmp(object->string, valid_keys[i]) == 0) {
        found = 1;
        break;
      }
    }
    if (!found) {
      printf("Unexpected setting in stream_action.so settings: %s\n",
        object->string);
      return 0;
    }
    object = object->next;
  }
  return 1;
}

// Takes a cJSON object and returns 1 if it's true, 0 if it's false, and -1 if
// it's invalid or not a boolean. Returns -1 if object is NULL.
static int GetCJSONBoolean(cJSON *object) {
  if (!object) return -1;
  if (object->type == cJSON_True) return 1;
  if (object->type == cJSON_False) return 0;
  return -1;
}

// Since this is such a long string of code, it gets moved into a separate
// function. Parses the parameters for a kernel action. Requires the cJSON
// *parameters* object for a kernel action, and fills in the KernelParameters.
// Returns 0 on error.
static int ParseKernelParameters(cJSON *json_parameters,
    KernelParameters *kernel_config,
    InitializationParameters *default_params) {
  cJSON *entry = NULL;
  // Due to the complexity of this config, this can forestall confusing errors
  // by pointing out misspelled keys.
  static const char* const valid_keys[] = {
    "type",
    "thread_count",
    "block_count",
    "shared_memory_size",
    "comment",
    "duration",
  };
  if (!VerifyJSONKeys(json_parameters, valid_keys, sizeof(valid_keys) /
    sizeof(char*))) {
    return 0;
  }
  // Determine whether the kernel should be a timer spin (constant time) or
  // counter spin (constant effort). The default is constant time, if the
  // setting isn't provided.
  entry = cJSON_GetObjectItem(json_parameters, "type");
  if (entry) {
    if (entry->type != cJSON_String) {
      printf("Invalid kernel type for kernel action.\n");
      return 0;
    }
    if (strcmp(entry->valuestring, "timer_spin") == 0) {
      kernel_config->use_counter_spin = 0;
    } else if (strcmp(entry->valuestring, "counter_spin") == 0) {
      kernel_config->use_counter_spin = 1;
    } else {
      printf("Unsupported kernel type for kernel action: %s\n",
        entry->valuestring);
      return 0;
    }
  } else {
    kernel_config->use_counter_spin = 0;
  }
  // Get the one required numerical parameter: duration.
  entry = cJSON_GetObjectItem(json_parameters, "duration");
  if (!entry || (entry->type != cJSON_Number)) {
    printf("Missing/invalid duration for kernel action.\n");
    return 0;
  }
  kernel_config->duration = (uint64_t) entry->valuedouble;
  // Get the block and thread counts, which default to the benchmark setting
  // if they aren't provided.
  if (!GetSingleBlockAndGridDimensions(default_params,
    &kernel_config->thread_count, &kernel_config->block_count)) {
    // No need to print a message on error, Get...Dimensions() does.
    return 0;
  }
  entry = cJSON_GetObjectItem(json_parameters, "block_count");
  if (entry) {
    if (entry->type != cJSON_Number) {
      printf("Invalid block count for kernel action.\n");
      return 0;
    }
    kernel_config->block_count = entry->valueint;
  }
  entry = cJSON_GetObjectItem(json_parameters, "thread_count");
  if (entry) {
    if (entry->type != cJSON_Number) {
      printf("Invalid thread count for kernel action.\n");
      return 0;
    }
    kernel_config->thread_count = entry->valueint;
  }
  // Unlike the other numbers, the shared_memory_count is optional and needs
  // validation.
  entry = cJSON_GetObjectItem(json_parameters, "shared_memory_size");
  if (entry) {
    if (entry->type != cJSON_Number) {
      printf("Invalid shared memory size for kernel action.\n");
      return 0;
    }
    kernel_config->shared_memory_count = entry->valueint;
  } else {
    kernel_config->shared_memory_count = 0;
  }
  switch (kernel_config->shared_memory_count) {
    case 0:
    case 4096:
    case 8192:
    case 10240:
      break;
    default:
      printf("Unsupported shared memory size for kernel action: %d\n",
        kernel_config->shared_memory_count);
      return 0;
  }
  return 1;
}

// Parses parameters for the malloc action. Returns 0 on error.
static int ParseMallocParameters(cJSON *json_parameters,
    MallocParameters *malloc_config) {
  cJSON *entry = NULL;
  int host = 0;
  static const char* const valid_keys[] = {
    "size",
    "host",
    "comment",
  };
  if (!VerifyJSONKeys(json_parameters, valid_keys, sizeof(valid_keys) /
    sizeof(char*))) {
    return 0;
  }
  entry = cJSON_GetObjectItem(json_parameters, "host");
  if (entry) {
    host = GetCJSONBoolean(entry);
  }
  if (host < 0) {
    printf("Invalid host setting for malloc action.\n");
    return 0;
  }
  malloc_config->allocate_host_memory = host;
  entry = cJSON_GetObjectItem(json_parameters, "size");
  if (!entry || (entry->type != cJSON_Number)) {
    printf("Missing/invalid size setting for malloc action.\n");
    return 0;
  }
  malloc_config->size = (uint64_t) entry->valuedouble;
  return 1;
}

// Parses the given (optional) parameters for the hipFree action. Returns 0
// on error. Since the parameters are optional, json_parameters can be NULL.
static int ParseFreeParameters(cJSON *json_parameters,
    FreeParameters *free_config) {
  cJSON *entry = NULL;
  int host = 0;
  static const char* const valid_keys[] = {
    "host",
    "comment",
  };
  // The config here is optional.
  free_config->free_host_memory = 0;
  if (!json_parameters) return 1;
  if (!VerifyJSONKeys(json_parameters, valid_keys, sizeof(valid_keys) /
    sizeof(char*))) {
    return 0;
  }
  entry = cJSON_GetObjectItem(json_parameters, "host");
  if (entry) {
    host = GetCJSONBoolean(entry);
  }
  if (host < 0) {
    printf("Invalid host setting for malloc action.\n");
    return 0;
  }
  free_config->free_host_memory = host;
  return 1;
}

// Parses JSON parameters for the memset action. Returns 0 on error.
static int ParseMemsetParameters(cJSON *json_parameters,
    MemsetParameters *memset_config) {
  cJSON *entry = NULL;
  int async = 1;
  static const char* const valid_keys[] = {
    "async",
    "size",
    "comment",
  };
  if (!VerifyJSONKeys(json_parameters, valid_keys, sizeof(valid_keys) /
    sizeof(char*))) {
    return 0;
  }
  entry = cJSON_GetObjectItem(json_parameters, "size");
  if (!entry || (entry->type != cJSON_Number)) {
    printf("Missing/invalid size for memset action.\n");
    return 0;
  }
  memset_config->size = (uint64_t) entry->valuedouble;
  entry = cJSON_GetObjectItem(json_parameters, "async");
  if (entry) {
    async = GetCJSONBoolean(entry);
  }
  if (async < 0) {
    printf("Invalid async setting for memset action.\n");
    return 0;
  }
  memset_config->synchronous = !async;
  return 1;
}

// Parses JSON parameters for the memcpy action. Returns 0 on error.
static int ParseMemcpyParameters(cJSON *json_parameters,
    MemcpyParameters *memcpy_config) {
  cJSON *entry = NULL;
  int async = 1;
  static const char* const valid_keys[] = {
    "async",
    "size",
    "direction",
    "comment",
  };
  if (!VerifyJSONKeys(json_parameters, valid_keys, sizeof(valid_keys) /
    sizeof(char*))) {
    return 0;
  }
  entry = cJSON_GetObjectItem(json_parameters, "size");
  if (!entry || (entry->type != cJSON_Number)) {
    printf("Missing/invalid size for memcpy action.\n");
    return 0;
  }
  memcpy_config->size = (uint64_t) entry->valuedouble;
  entry = cJSON_GetObjectItem(json_parameters, "async");
  if (entry) {
    async = GetCJSONBoolean(entry);
  }
  if (async < 0) {
    printf("Invalid async setting for memcpy action.\n");
    return 0;
  }
  memcpy_config->synchronous = !async;
  entry = cJSON_GetObjectItem(json_parameters, "direction");
  if (!entry || (entry->type != cJSON_String)) {
    printf("Missing/invalid direction for memcpy action.\n");
    return 0;
  }
  if (strcmp(entry->valuestring, "deviceToHost") == 0) {
    memcpy_config->direction = hipMemcpyDeviceToHost;
  } else if (strcmp(entry->valuestring, "hostToDevice") == 0) {
    memcpy_config->direction = hipMemcpyHostToDevice;
  } else if (strcmp(entry->valuestring, "deviceToDevice") == 0) {
    memcpy_config->direction = hipMemcpyDeviceToDevice;
  } else {
    printf("Unsupported direction for memcpy action: %s\n",
      entry->valuestring);
    return 0;
  }
  return 1;
}

// Parses the JSON parameters for the "synchronize" action. Returns 0 on error.
// The json_parameters can be NULL, in which case the sync parameters will take
// their default values.
static int ParseSyncParameters(cJSON *json_parameters,
    SyncParameters *sync_config) {
  cJSON *entry = NULL;
  static const char* const valid_keys[] = {
    "device",
    "comment",
  };
  sync_config->sync_device = 0;
  if (!json_parameters) return 1;
  if (!VerifyJSONKeys(json_parameters, valid_keys, sizeof(valid_keys) /
    sizeof(char*))) {
    return 0;
  }
  entry = cJSON_GetObjectItem(json_parameters, "device");
  if (entry) {
    sync_config->sync_device = GetCJSONBoolean(entry);
  }
  if (sync_config->sync_device < 0) {
    sync_config->sync_device = 0;
    printf("Invalid device setting for sync action.\n");
    return 0;
  }
  return 1;
}

// Parses a JSON action object in order to fill in the given ActionConfig.
// Returns 0 on error and 1 on success. May partially initialize action on
// error, so the caller may need to clean it up. However, the action type is
// guaranteed to be valid if any other fields are set.
static int ParseSingleAction(cJSON *object, ActionConfig *action,
    InitializationParameters *params) {
  cJSON *entry = NULL;
  ActionType type = ACTION_UNINITIALIZED;
  static const char* const valid_keys[] = {
    "type",
    "label",
    "delay",
    "parameters",
    "comment",
  };
  // Validate keys to find confusing spelling mistakes that may make a setting
  // take its default value unintentionally.
  if (!VerifyJSONKeys(object, valid_keys, sizeof(valid_keys) /
    sizeof(char*))) {
    return 0;
  }
  // Start with the hardest property: the action's type.
  entry = cJSON_GetObjectItem(object, "type");
  if (!entry || (entry->type != cJSON_String)) {
    printf("Missing/invalid action type for stream_action.so.\n");
    return 0;
  }
  if (strcmp(entry->valuestring, "kernel") == 0) {
    type = ACTION_KERNEL;
  } else if (strcmp(entry->valuestring, "malloc") == 0) {
    type = ACTION_MALLOC;
  } else if (strcmp(entry->valuestring, "free") == 0) {
    type = ACTION_FREE;
  } else if (strcmp(entry->valuestring, "memset") == 0) {
    type = ACTION_MEMSET;
  } else if (strcmp(entry->valuestring, "memcpy") == 0) {
    type = ACTION_MEMCPY;
  } else if (strcmp(entry->valuestring, "synchronize") == 0) {
    type = ACTION_SYNC;
  } else {
    printf("Unsupported action type for stream_action.so: %s\n",
      entry->valuestring);
    return 0;
  }
  action->type = type;
  entry = cJSON_GetObjectItem(object, "label");
  if (!entry || (entry->type != cJSON_String)) {
    printf("Missing/invalid action label for stream_action.so.\n");
    return 0;
  }
  action->label = strdup(entry->valuestring);
  if (!action->label) return 0;
  entry = cJSON_GetObjectItem(object, "delay");
  if (entry) {
    if (entry->type != cJSON_Number) {
      printf("Invalid delay for stream_action.so.\n");
      return 0;
    }
    action->delay = entry->valuedouble;
  }
  // Last, parse the action-specific parameters. Remember that additional
  // parameters are optional for some actions, so only ensure that the
  // parameters are an object if they're non-NULL.
  entry = cJSON_GetObjectItem(object, "parameters");
  if (entry && (entry->type != cJSON_Object)) {
    printf("Invalid action parameters for stream_action.so.\n");
    return 0;
  }
  // Get kernel config parsing over with first, since it's the most complex.
  if (type == ACTION_KERNEL) {
    if (!entry) {
      printf("Missing kernel parameters for stream_action.so.\n");
      return 0;
    }
    if (!ParseKernelParameters(entry, &(action->parameters.kernel), params)) {
      return 0;
    }
  }
  if (type == ACTION_MALLOC) {
    if (!entry) {
      printf("Missing malloc parameters for stream_action.so.\n");
      return 0;
    }
    if (!ParseMallocParameters(entry, &(action->parameters.malloc))) return 0;
  }
  if (type == ACTION_FREE) {
    // It's okay for "entry" to be NULL here.
    if (!ParseFreeParameters(entry, &(action->parameters.free))) return 0;
  }
  if (type == ACTION_MEMSET) {
    if (!entry) {
      printf("Missing memset parameters for stream_action.so.\n");
      return 0;
    }
    if (!ParseMemsetParameters(entry, &(action->parameters.memset))) return 0;
  }
  if (type == ACTION_MEMCPY) {
    if (!entry) {
      printf("Missing memcpy parameters for stream_action.so.\n");
      return 0;
    }
    if (!ParseMemcpyParameters(entry, &(action->parameters.memcpy))) return 0;
  }
  if (type == ACTION_SYNC) {
    // It's okay for "entry" to be NULL here, too.
    if (!ParseSyncParameters(entry, &(action->parameters.sync))) return 0;
  }
  return 1;
}

// Takes a TaskState struct to be initialized and a JSON configuration string.
// Parses the JSON configuration and fills the appropriate fields in the state
// struct. The stream_priority value is needed, because this function will
// create the CUDA stream if the use_null_stream setting is not true. Returns 0
// on error.
static int ParseParameters(TaskState *state,
    InitializationParameters *params) {
  cJSON *json_root = NULL;
  cJSON *list_head = NULL;
  cJSON *entry = NULL;
  ActionConfig *actions = NULL;
  ActionConfig *action = NULL;
  int i = 0, action_count = 0, use_null_stream = 0;
  static const char* const valid_keys[] = {
    "actions",
    "use_null_stream",
    "comment",
  };
  json_root = cJSON_Parse(params->additional_info);
  if (!json_root || (json_root->type != cJSON_Object)) {
    printf("Missing/invalid additional_info for stream_action.so.\n");
    goto ErrorCleanup;
  }
  if (!VerifyJSONKeys(json_root, valid_keys, sizeof(valid_keys) /
    sizeof(char*))) {
    goto ErrorCleanup;
  }
  // First, check for the "use_null_stream" setting.
  entry = cJSON_GetObjectItem(json_root, "use_null_stream");
  if (entry) use_null_stream = GetCJSONBoolean(entry);
  if (use_null_stream < 0) {
    printf("Invalid use_null_stream setting in stream_action.so.\n");
    goto ErrorCleanup;
  }
  // Always use a user-defined stream for copy_out operations.
  if (!CheckCUDAError(CreateCUDAStreamWithPriorityAndMask(
    params->stream_priority, 0, &(state->copy_out_stream)))) {
    goto ErrorCleanup;
  }
  state->stream_created = 1;
  // If the NULL stream wasn't specified, then use the user-defined stream
  // for all other operations, too.
  if (use_null_stream) {
    state->stream = hipStreamLegacy;
  } else {
    state->stream = state->copy_out_stream;
  }
  // Get the actions list, ensuring it's an array with at least one element.
  list_head = cJSON_GetObjectItem(json_root, "actions");
  if (!list_head || (list_head->type != cJSON_Array) || !list_head->child) {
    printf("Missing/invalid list of actions for stream_action.so.\n");
    goto ErrorCleanup;
  }
  // Count the number of actions in the list.
  entry = list_head->child;
  action_count = 1;
  while (entry->next) {
    action_count++;
    entry = entry->next;
  }
  // Allocate and initialize the internal list of ActionConfig structs.
  actions = (ActionConfig *) calloc(action_count, sizeof(*actions));
  if (!actions) goto ErrorCleanup;
  entry = list_head->child;
  for (i = 0; i < action_count; i++) {
    action = actions + i;
    if (!ParseSingleAction(entry, action, params)) goto ErrorCleanup;
    entry = entry->next;
  }
  // Clean up and return success.
  state->actions = actions;
  state->action_count = action_count;
  cJSON_Delete(json_root);
  return 1;
ErrorCleanup:
  if (json_root) cJSON_Delete(json_root);
  if (actions) {
    for (i = 0; i < action_count; i++) {
      CleanupAction(actions + i);
    }
    free(actions);
  }
  return 0;
}

// Allocates buffers needed by a single kernel action. Returns 0 on error.
static int AllocateKernelActionMemory(KernelParameters *parameters) {
  size_t block_times_size = 2 * parameters->block_count * sizeof(uint64_t);
  size_t smids_size = parameters->block_count * sizeof(uint32_t);
  if (!CheckCUDAError(hipMalloc(&parameters->device_block_times,
    block_times_size))) {
    return 0;
  }
  if (!CheckCUDAError(hipMalloc(&parameters->device_smids, smids_size))) {
    return 0;
  }
  if (!CheckCUDAError(hipHostMalloc(&parameters->host_block_times,
    block_times_size))) {
    return 0;
  }
  if (!CheckCUDAError(hipHostMalloc(&parameters->host_smids, smids_size))) {
    return 0;
  }
  return 1;
}

// Preallocates a set of buffers so that a limited number of free actions don't
// necessarily need to follow malloc actions. Returns 0 on error.
static int PreallocateFreeActionBuffers(TaskState *state) {
  int i;
  uint8_t **dest = NULL;
  for (i = 0; i < INITIAL_ALLOCATION_COUNT; i++) {
    dest = state->device_memory_allocations + i;
    if (!CheckCUDAError(hipMalloc(dest, INITIAL_ALLOCATION_SIZE))) {
      return 0;
    }
    // Increment these values one step at a time, so they can be cleaned up
    // properly if one of the later allocations fails.
    state->device_memory_allocation_count++;
    dest = state->host_memory_allocations + i;
    if (!CheckCUDAError(hipHostMalloc(dest, INITIAL_ALLOCATION_SIZE))) {
      return 0;
    }
    state->host_memory_allocation_count++;
  }
  return 1;
}

// Takes a TaskState after fully parsing InitializationParameters (i.e. the
// actions list is populated). Allocates necessary buffers for kernel actions,
// memory sets and copies, holding pointers for malloc actions, and buffers of
// data to report to the calling process during copy_out. Returns 0 on error.
static int AllocateMemory(TaskState *state) {
  int i;
  uint64_t current_size;
  uint64_t max_size = 0;
  int secondary_buffer_needed = 0;
  int malloc_action_exists = 0;
  int kernel_count = 0;
  ActionConfig *action = NULL;
  // Collect aggregate information about all actions, and allocate the kernel
  // action's buffers while we're at it.
  for (i = 0; i < state->action_count; i++) {
    action = state->actions + i;
    switch (action->type) {
      case ACTION_KERNEL:
        kernel_count++;
        if (!AllocateKernelActionMemory(&(action->parameters.kernel))) {
          return 0;
        }
        break;
      case ACTION_MALLOC:
        malloc_action_exists = 1;
        break;
      case ACTION_FREE:
        malloc_action_exists = 1;
      case ACTION_MEMSET:
        current_size = action->parameters.memset.size;
        if (current_size > max_size) max_size = current_size;
        break;
      case ACTION_MEMCPY:
        current_size = action->parameters.memcpy.size;
        if (current_size > max_size) max_size = current_size;
        if (action->parameters.memcpy.direction == hipMemcpyDeviceToDevice) {
          secondary_buffer_needed = 1;
        }
        break;
      default:
        break;
    }
  }
  // Start by allocating device memory.
  if (!CheckCUDAError(hipMalloc(&state->device_copy_buffer, max_size))) {
    return 0;
  }
  // Only allocate a second device buffer if a device-to-device memcpy action
  // is present.
  if (secondary_buffer_needed) {
    if (!CheckCUDAError(hipMalloc(&state->device_secondary_buffer,
      max_size))) {
      return 0;
    }
  }
  // Now allocate host memory.
  if (!CheckCUDAError(hipHostMalloc(&state->host_copy_buffer, max_size))) {
    return 0;
  }
  if (malloc_action_exists) {
    state->device_memory_allocations = (uint8_t**) calloc(
      MAX_MEMORY_ALLOCATION_COUNT, sizeof(uint8_t*));
    if (!state->device_memory_allocations) {
      printf("Failed allocating list of device memory allocation pointers.\n");
      return 0;
    }
    state->host_memory_allocations = (uint8_t**) calloc(
      MAX_MEMORY_ALLOCATION_COUNT, sizeof(uint8_t*));
    if (!state->host_memory_allocations) {
      printf("Failed allocating list of host memory allocation pointers.\n");
      return 0;
    }
    if (!PreallocateFreeActionBuffers(state)) return 0;
  }
  // Any pointers contained in the individual KernelTimes entries are simply
  // copied from KernelParameters structs after execution--they don't need to
  // be allocated here.
  state->kernel_times = (KernelTimes*) calloc(kernel_count,
    sizeof(KernelTimes));
  if (!state->kernel_times) {
    printf("Failed allocating list of kernel times.\n");
    return 0;
  }
  state->kernel_count = kernel_count;
  return 1;
}

// Initializes the tasks' kernel_times array. Must be called after memory
// allocation. This is done once because most of the fields in the kernel_times
// array never change, apart from cuda_launch_times. Returns 0 on error.
static int InitializeKernelTimes(TaskState *state) {
  int i;
  int kernel_index = 0;
  KernelTimes *current_times = NULL;
  ActionConfig *action = NULL;
  KernelParameters *params = NULL;
  for (i = 0; i < state->action_count; i++) {
    action = state->actions + i;
    if (action->type != ACTION_KERNEL) continue;
    params = &(action->parameters.kernel);
    current_times = state->kernel_times + kernel_index;
    current_times->kernel_name = action->label;
    current_times->block_count = params->block_count;
    current_times->thread_count = params->thread_count;
    current_times->shared_memory = params->shared_memory_count * 4;
    current_times->block_times = params->host_block_times;
    current_times->block_smids = params->host_smids;
    kernel_index++;
  }
  return 1;
}

static void* Initialize(InitializationParameters *params) {
  TaskState *state = NULL;
  state = (TaskState *) malloc(sizeof(*state));
  if (!state) {
    printf("Error allocating memory for stream_action task state.\n");
    return NULL;
  }
  memset(state, 0, sizeof(*state));
  if (!CheckCUDAError(hipSetDevice(params->cuda_device))) {
    Cleanup(state);
    return NULL;
  }
  // Parse the configuration string, initialize the action configs, and create
  // the CUDA stream (if a non-NULL stream is used).
  if (!ParseParameters(state, params)) {
    Cleanup(state);
    return NULL;
  }
  if (!AllocateMemory(state)) {
    Cleanup(state);
    return NULL;
  }
  if (!InitializeKernelTimes(state)) {
    Cleanup(state);
    return NULL;
  }
  return state;
}

// Nothing needs to be copied to the GPU at this stage in the benchmark.
static int CopyIn(void *data) {
  return 1;
}

// Copies device data to host buffers for a single kernel action. Requires a
// stream on which the copy should run. Returns 0 on error.
static int CopyKernelActionMemoryOut(KernelParameters *kernel,
    hipStream_t stream) {
  size_t block_times_size = 2 * kernel->block_count * sizeof(uint64_t);
  size_t block_smids_size = kernel->block_count * sizeof(uint32_t);
  if (!CheckCUDAError(hipMemcpyAsync(kernel->host_block_times,
    kernel->device_block_times, block_times_size, hipMemcpyDeviceToHost,
    stream))) {
    return 0;
  }
  if (!CheckCUDAError(hipMemcpyAsync(kernel->host_smids, kernel->device_smids,
    block_smids_size, hipMemcpyDeviceToHost, stream))) {
    return 0;
  }
  return 1;
}

// Provides the caller with information about the kernel actions.
static int CopyOut(void *data, TimingInformation *times) {
  TaskState *state = (TaskState *) data;
  int i;
  for (i = 0; i < state->action_count; i++) {
    if (state->actions[i].type != ACTION_KERNEL) continue;
    if (!CopyKernelActionMemoryOut(&(state->actions[i].parameters.kernel),
      state->copy_out_stream)) {
      return 0;
    }
  }
  if (!CheckCUDAError(hipStreamSynchronize(state->copy_out_stream))) return 0;
  // The kernel_times structs were already filled in with the correct pointers
  // during initialization, and the cuda_launch_times were filled in during the
  // execute phase. So now, all that needs to be done is provide the correct
  // pointer.
  times->kernel_count = state->kernel_count;
  times->kernel_info = state->kernel_times;
  times->resulting_data_size = 0;
  times->resulting_data = NULL;
  return 1;
}

// Executes a kernel action. Requires the index of the kernel action (its order
// relative only to other kernel actions) in order to fill in the CUDA launch
// times in the correct entry in the kernel_times array. Returns 0 on error.
static int ExecuteKernelAction(TaskState *state, KernelParameters *params,
    int kernel_index) {
  KernelTimes *kernel_time = state->kernel_times + kernel_index;
  kernel_time->cuda_launch_times[0] = CurrentSeconds();
  switch (params->shared_memory_count) {
    case 0:
      GPUSpin<<<params->block_count, params->thread_count, 0, state->stream>>>(
        params->use_counter_spin, params->duration, params->device_block_times,
        params->device_smids, NULL);
      break;
    case 4096:
      SharedMemGPUSpin_4096<<<params->block_count, params->thread_count, 0,
        state->stream>>>(params->use_counter_spin, params->duration,
        params->device_block_times, params->device_smids, NULL);
      break;
    case 8192:
      SharedMemGPUSpin_8192<<<params->block_count, params->thread_count, 0,
        state->stream>>>(params->use_counter_spin, params->duration,
        params->device_block_times, params->device_smids, NULL);
      break;
    case 10240:
      SharedMemGPUSpin_10240<<<params->block_count, params->thread_count, 0,
        state->stream>>>(params->use_counter_spin, params->duration,
        params->device_block_times, params->device_smids, NULL);
      break;
    default:
      printf("Unsupported kernel shared memory count: %d\n",
        params->shared_memory_count);
      return 0;
  }
  // Record the time after the kernel launch returns, but we don't know when
  // synchronization will complete in this benchmark, so set that entry to 0.
  kernel_time->cuda_launch_times[1] = CurrentSeconds();
  kernel_time->cuda_launch_times[2] = 0;
  return 1;
}

// Executes a malloc action. Returns 0 on error.
static int ExecuteMallocAction(TaskState *state, MallocParameters *params) {
  int next_index = 0;
  uint8_t **destination = NULL;
  if (params->allocate_host_memory) {
    next_index = state->host_memory_allocation_count;
  } else {
    next_index = state->device_memory_allocation_count;
  }
  if (next_index >= MAX_MEMORY_ALLOCATION_COUNT) {
    printf("Can't execute malloc action: too many unfreed %s allocations.\n",
      params->allocate_host_memory ? "host" : "device");
    return 0;
  }
  if (params->allocate_host_memory) {
    destination = state->host_memory_allocations + next_index;
    if (!CheckCUDAError(hipHostMalloc(destination, params->size))) return 0;
    state->host_memory_allocation_count++;
    return 1;
  }
  destination = state->device_memory_allocations + next_index;
  if (!CheckCUDAError(hipMalloc(destination, params->size))) return 0;
  state->device_memory_allocation_count++;
  return 1;
}

// Executes a free action. Returns 0 on error.
static int ExecuteFreeAction(TaskState *state, FreeParameters *params) {
  if (params->free_host_memory) {
    if (state->host_memory_allocation_count == 0) {
      printf("Can't execute free action: No host memory allocations.\n");
      return 0;
    }
    state->host_memory_allocation_count--;
    if (!CheckCUDAError(hipHostFree(state->host_memory_allocations[
      state->host_memory_allocation_count]))) {
      return 0;
    }
    return 1;
  }
  if (state->device_memory_allocation_count == 0) {
    printf("Can't execute free action: No device memory allocations.\n");
    return 0;
  }
  state->device_memory_allocation_count--;
  if (!CheckCUDAError(hipFree(state->device_memory_allocations[
    state->device_memory_allocation_count]))) {
    return 0;
  }
  return 1;
}

// Executes a memset action. Fills a device buffer with a random value. Returns
// 0 on error.
static int ExecuteMemsetAction(TaskState *state, MemsetParameters *params) {
  if (params->synchronous) {
    if (!CheckCUDAError(hipMemset(state->device_copy_buffer, rand(),
      params->size))) {
      return 0;
    }
    return 1;
  }
  if (!CheckCUDAError(hipMemsetAsync(state->device_copy_buffer, rand(),
    params->size, state->stream))) {
    return 0;
  }
  return 1;
}

// Executes a memcpy action. Returns 0 on error.
static int ExecuteMemcpyAction(TaskState *state, MemcpyParameters *params) {
  uint8_t *src = NULL;
  uint8_t *dest = NULL;
  switch (params->direction) {
    case hipMemcpyDeviceToDevice:
      src = state->device_copy_buffer;
      dest = state->device_secondary_buffer;
      break;
    case hipMemcpyDeviceToHost:
      src = state->device_copy_buffer;
      dest = state->host_copy_buffer;
      break;
    case hipMemcpyHostToDevice:
      src = state->host_copy_buffer;
      dest = state->device_copy_buffer;
      break;
    default:
      printf("Unsupported direction for memcpy action: %d\n",
        (int) params->direction);
      return 0;
  }
  if (params->synchronous) {
    if (!CheckCUDAError(hipMemcpy(dest, src, params->size,
      params->direction))) {
      return 0;
    }
    return 1;
  }
  if (!CheckCUDAError(hipMemcpyAsync(dest, src, params->size,
    params->direction, state->stream))) {
    return 0;
  }
  return 1;
}

// Executes a synchronization action. Returns 0 on error.
static int ExecuteSyncAction(TaskState *state, SyncParameters *params) {
  if (params->sync_device) {
    if (!CheckCUDAError(hipDeviceSynchronize())) return 0;
    return 1;
  }
  if (!CheckCUDAError(hipStreamSynchronize(state->stream))) return 0;
  return 1;
}

// Sleeps for at least the given number of seconds, with a microsecond
// granularity.
static void SleepSeconds(double seconds) {
  uint64_t to_sleep = (uint64_t) (seconds * 1e6);
  usleep(to_sleep);
}

// Executes each action in the order it appears in the list.
static int Execute(void *data) {
  TaskState *state = (TaskState *) data;
  ActionConfig *action = NULL;
  int kernel_index = 0;
  int i;
  for (i = 0; i < state->action_count; i++) {
    action = state->actions + i;
    if (action->delay > 0.0) {
      SleepSeconds(state->actions[i].delay);
    }
    switch (action->type) {
      case ACTION_KERNEL:
        if (!ExecuteKernelAction(state, &(action->parameters.kernel),
          kernel_index)) {
          return 0;
        }
        kernel_index++;
        break;
      case ACTION_MALLOC:
        if (!ExecuteMallocAction(state, &(action->parameters.malloc))) {
          return 0;
        }
        break;
      case ACTION_FREE:
        if (!ExecuteFreeAction(state, &(action->parameters.free))) {
          return 0;
        }
        break;
      case ACTION_MEMSET:
        if (!ExecuteMemsetAction(state, &(action->parameters.memset))) {
          return 0;
        }
        break;
      case ACTION_MEMCPY:
        if (!ExecuteMemcpyAction(state, &(action->parameters.memcpy))) {
          return 0;
        }
        break;
      case ACTION_SYNC:
        if (!ExecuteSyncAction(state, &(action->parameters.sync))) {
          return 0;
        }
        break;
      default:
        printf("Attempted to execute invalid action: %d\n", action->type);
        return 0;
    }
  }
  if (!CheckCUDAError(hipStreamSynchronize(state->stream))) return 0;
  return 1;
}

static const char* GetName(void) {
  return "Sequential action execution";
}

int RegisterFunctions(BenchmarkLibraryFunctions *functions) {
  functions->initialize = Initialize;
  functions->copy_in = CopyIn;
  functions->execute = Execute;
  functions->copy_out = CopyOut;
  functions->cleanup = Cleanup;
  functions->get_name = GetName;
  return 1;
}
