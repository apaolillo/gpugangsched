#include "hip/hip_runtime.h"
#include <codecvt>
#include <cstdint>
#include <iostream>
#include <hip/hip_runtime.h>
#include "../benchmark_gpu_utilities.h"
#include "../library_interface.h"
#include "../benchmark_gpu_utilities.h"
#include "libsmctrl/libsmctrl.h"
#include "../third_party/cJSON.h"




// Holds the local state for one instance of this benchmark.
typedef struct {
  // The CUDA stream with which all operations will be associated.
  hipStream_t stream;
  int stream_created;
  // Holds the device copy of the start and end times of each block.
  uint64_t *device_block_times;
  // Holds the device copy of the SMID each block was assigned to.
  uint32_t *device_block_smids;
  

  //buffer that stores the result of the loop
  float *device_buffer;
  
  //mask used for the stream.
  uint64_t smMask;

  int block_count;
  int thread_count;
  // Holds host-side times that are shared with the calling process.
  KernelTimes busy_kernel_times;
  
} TaskState;





// Implements the cleanup function required by the library interface, but is
// also called internally (only during Initialize()) to clean up after errors.
static void Cleanup(void *data) {
  TaskState *state = (TaskState *) data;
  KernelTimes *host_times = &state->busy_kernel_times;
  // Free device memory.
  if (state->device_block_times) hipFree(state->device_block_times);
  if (state->device_block_smids) hipFree(state->device_block_smids);
  if(state->device_buffer) hipFree(state->device_buffer);
  // Free host memory.
  if (host_times->block_times) hipHostFree(host_times->block_times);
  if (host_times->block_smids) hipHostFree(host_times->block_smids);
  if (state->stream_created) {
    // Call CheckCUDAError here to print a message, even though we won't check
    // the return value.
    CheckCUDAError(hipStreamDestroy(state->stream));
  }



  memset(state, 0, sizeof(*state));
  free(state);
}


// Allocates GPU and CPU memory. Returns 0 on error, 1 otherwise.
static int AllocateMemory(TaskState *state) {
  uint64_t block_times_size = state->block_count * sizeof(uint64_t) * 2;
  uint64_t block_smids_size = state->block_count * sizeof(uint32_t);
  KernelTimes *host_times = &state->busy_kernel_times;

  // Allocate device memory
  if (!CheckCUDAError(hipMalloc(&(state->device_block_times),
    block_times_size))) {
    return 0;
  }
  if (!CheckCUDAError(hipMalloc(&(state->device_block_smids),
    block_smids_size))) {
    return 0;
  }
  if(!CheckCUDAError(hipMalloc(&(state->device_buffer), sizeof(float) * 1000000))){
    return 0;
  }
  // Allocate host memory.
  if (!CheckCUDAError(hipHostMalloc(&host_times->block_times,
    block_times_size))) {
    return 0;
  }
  if (!CheckCUDAError(hipHostMalloc(&host_times->block_smids,
    block_smids_size))) {
    return 0;
  }
  return 1;
}

static int SetMask(TaskState *state){
  
  //create the stream using the stream pointer from the state struct. 
  if(!CheckCUDAError(hipStreamCreate(&state->stream))){
    std::cout<<"failed to create stream\n";
    return 0;
  };

  //if a mask was provided, set it.
  if(state->smMask){
    libsmctrl_set_stream_mask(state->stream, state->smMask);
  }
  return 1;
}

static int initKernelConfigs(TaskState *state, char * info){
  cJSON *parsed = NULL;
  cJSON *entry = NULL;
  cJSON *list_entry = NULL;

//parse the config file to get the mask in the additional info section.
//If no mask is given, return from the function without parsing the remaining part of the json file. 
 parsed = cJSON_Parse(info);
  if (!parsed || (parsed->type != cJSON_Array) || !parsed->child) {
    printf("no mask given for kernel launch\n");
    return 1;
  }

  list_entry = parsed->child;
  
  entry = cJSON_GetObjectItem(list_entry, "sm_mask");
  

  if(entry){
    //get the sm mask value and set it.
    uint64_t mask = strtoull(entry->valuestring, NULL, 16);
    std::cout<<mask<<"\t\n";
    state->smMask = mask;
  }

  return 1;

}

static void* Initialize(InitializationParameters *params) {
  TaskState *state = NULL;
 // First allocate space for local data.
  state = (TaskState *) calloc(1, sizeof(*state));
  if (!state) return NULL;
  if (!CheckCUDAError(hipSetDevice(params->cuda_device))) return NULL;
  if (!GetSingleBlockAndGridDimensions(params, &state->thread_count,
    &state->block_count)) {
    Cleanup(state);
    return NULL;
  }
  if (!AllocateMemory(state)) {
    Cleanup(state);
    return NULL;
  }
  if(!initKernelConfigs(state, params->additional_info)){
    Cleanup(state);
    return NULL;
  }
  if (!(SetMask(state))) {
    Cleanup(state);
    return NULL;
  }
  
  state->stream_created = 1;

  return state;
}


// Nothing needs to be copied in for this benchmark.
static int CopyIn(void *data) {
  return 1;
}


// Kernel that performs a simple computation
static __global__ void maxUtilizationKernel(float* output, int n, uint64_t *block_times, uint32_t *block_smids)
{

  uint64_t start_time = GlobalTimer64();
  // First, record the kernel and block start times
  if (threadIdx.x == 0) {
    block_times[blockIdx.x * 2] = start_time;
    block_smids[blockIdx.x] = GetSMID();
  }
  __syncthreads();  
  
  //actual work of the kernel
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < n) {
      // Some computations to keep the SM busy
      float value = 0.0f;
      for (int i = 0; i < n; i++) {
          value += sinf(tid * 0.1f + i) * cosf(tid * 0.1f);
      }
      output[tid] = value;
  }
  // Record the kernel and block end times.
  if (threadIdx.x == 0) {
    block_times[blockIdx.x * 2 + 1] = GlobalTimer64();
  }
}





static int Execute(void *data) {
  TaskState *state = (TaskState *) data;
  state->busy_kernel_times.cuda_launch_times[0] = CurrentSeconds();

  //launch the kernel. 
  maxUtilizationKernel<<<state->block_count, state->thread_count, 0, state->stream>>>(state->device_buffer, 200000, state->device_block_times, state->device_block_smids);

  state->busy_kernel_times.cuda_launch_times[1] = CurrentSeconds();
  if (!CheckCUDAError(hipStreamSynchronize(state->stream))) return 0;
  state->busy_kernel_times.cuda_launch_times[2] = CurrentSeconds();
  return 1;
}

static int CopyOut(void *data, TimingInformation *times) {
  TaskState *state = (TaskState *) data;
  KernelTimes *host_times = &state->busy_kernel_times;
  uint64_t block_times_count = state->block_count * 2;
  uint64_t block_smids_count = state->block_count;
  memset(times, 0, sizeof(*times));
  if (!CheckCUDAError(hipMemcpyAsync(host_times->block_times,
    state->device_block_times, block_times_count * sizeof(uint64_t),
    hipMemcpyDeviceToHost, state->stream))) {
    return 0;
  }
  if (!CheckCUDAError(hipMemcpyAsync(host_times->block_smids,
    state->device_block_smids, block_smids_count * sizeof(uint32_t),
    hipMemcpyDeviceToHost, state->stream))) {
    return 0;
  }
  if (!CheckCUDAError(hipStreamSynchronize(state->stream))) return 0;
  host_times->kernel_name = "busy Kernel";
  host_times->block_count = state->block_count;
  host_times->thread_count = state->thread_count;
  times->kernel_count = 1;
  times->kernel_info = host_times;
  return 1;
}

static const char* GetName(void) {
  return "Busy Kernel";
}

// This should be the only function we export from the library, to provide
// pointers to all of the other functions.
int RegisterFunctions(BenchmarkLibraryFunctions *functions) {
  functions->initialize = Initialize;
  functions->copy_in = CopyIn;
  functions->execute = Execute;
  functions->copy_out = CopyOut;
  functions->cleanup = Cleanup;
  functions->get_name = GetName;
  return 1;
}

