#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "libsmctrl.h"

// Kernel that performs a simple computation
__global__ void maxUtilizationKernel(float* output, int n)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) {
        // Some computations to keep the SM busy
        float value = 0.0f;
        for (int i = 0; i < 1000; i++) {
            value += sinf(tid * 0.1f + i) * cosf(tid * 0.1f);
        }
        output[tid] = value;
    }
}

int main()
{
    // Get device properties
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    std::cout<<"Numebr of SMs: "<<deviceProp.multiProcessorCount << "\n";
    
    hipStream_t busyStream;
    hipStreamCreate(&busyStream);

      //set a mask for the stream
    libsmctrl_set_stream_mask((void*)busyStream, 0xFFFFFFFFFFFFFF00);
    // Choose grid and block sizes to maximize SM utilization
    const int threadsPerBlock = 256;
    const int blocksPerSM = 32; // RTX 4070 Ti can handle multiple blocks per SM
    const int totalBlocks = deviceProp.multiProcessorCount * blocksPerSM;
    
    const int n = totalBlocks * threadsPerBlock;


    std::cout<<"Launching kernel with "<<totalBlocks<<" blocks, "<<threadsPerBlock<<" threads per block ("<<n<<" total threads)\n";

    
    // Allocate memory
    float* d_output;
    hipMalloc(&d_output, n * sizeof(float));
    
    // Launch kernel
    maxUtilizationKernel<<<totalBlocks, threadsPerBlock, 0, busyStream>>>(d_output, n);
    
    //change the mask between kernel launches. 
    libsmctrl_set_stream_mask((void*)busyStream, 0x00000000000000FF);
    
    //launch the same kernel but with a different mask for the stream. 
    maxUtilizationKernel<<<totalBlocks, threadsPerBlock, 0, busyStream>>>(d_output, n);

    // Wait for the kernel to finish
    hipDeviceSynchronize();
    
    // Clean up
    hipFree(d_output);
    hipDeviceReset();
    
    printf("Kernel execution completed\n");
    
    return 0;
}
